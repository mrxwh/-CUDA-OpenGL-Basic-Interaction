#include "hip/hip_runtime.h"
#include "kernel.h"
#include "stdio.h"
#include "math.h"
#include ""

#define TX 32
#define TY 32

struct GpuData {
    int w;
    int h;
    int* distanceData;
};

// 3d version
const dim3 blockSize(TX, TY, 1);
dim3 gridSize;

struct GpuData* gpuDataCPU; // used for freeing all cuda memory again
__device__
struct GpuData* gpuDataLOCAL; // do not use in device code!!!

__device__
unsigned char clip(int n) { return n > 255 ? 255 : (n < 0 ? 0 : n); }


__global__
void distanceKernel(GpuData* gpudata, int2 pos) {
    const int x = blockIdx.x * blockDim.x + threadIdx.x;
    const int y = blockIdx.y * blockDim.y + threadIdx.y;
    if ((x >= gpudata->w) || (y >= gpudata->h)) return; // Check if within image bounds
    const int i = x + y * gpudata->w; // 1D indexing
    const int dist = sqrtf((x - pos.x) * (x - pos.x) +
                           (y - pos.y) * (y - pos.y));
    gpudata->distanceData[i] = dist;
}

__global__
void imageKernel(uchar4 *d_out, GpuData* gpudata) {
    const int x = blockIdx.x * blockDim.x + threadIdx.x;
    const int y = blockIdx.y * blockDim.y + threadIdx.y;
    if ((x >= gpudata->w) || (y >= gpudata->h)) return; // Check if within image bounds
    const int i = x + y * gpudata->w; // 1D indexing
    const int dist = gpudata->distanceData[i];
    const unsigned char intensity = clip(255 - dist);
    d_out[i].x = intensity;
    d_out[i].y = intensity;
    d_out[i].z = 0;
    d_out[i].w = 255;
}

void kernelLauncher(uchar4 *d_out, int2 pos) {
    

    distanceKernel<<<gridSize, blockSize>>>(gpuDataLOCAL, pos);
    gpuErrchk(hipPeekAtLastError());
    imageKernel<<<gridSize, blockSize>>>(d_out, gpuDataLOCAL);
    gpuErrchk(hipPeekAtLastError());
}

void init(int w, int h) {

    gridSize = dim3((w + blockSize.x - 1) / blockSize.x, (h + blockSize.y - 1) / blockSize.y,
        1); // + TX - 1 for w size that is not divisible by TX
    // alloc cpu version of struct
    //GpuData tmp = { w,h };
    gpuDataCPU = (GpuData*)malloc(sizeof(GpuData));
    gpuDataCPU->w = w;
    gpuDataCPU->h = h;
    // fill it with cuda references
    hipMalloc((void**)&gpuDataCPU->distanceData, gpuDataCPU->w * gpuDataCPU->h * sizeof(*gpuDataCPU->distanceData));
    // alloc cuda version of struct
    hipMalloc((void**)&gpuDataLOCAL, sizeof(GpuData)); gpuErrchk(hipPeekAtLastError());
    // copy references to cuda version
    hipMemcpy(gpuDataLOCAL, gpuDataCPU, sizeof(GpuData), hipMemcpyHostToDevice); gpuErrchk(hipPeekAtLastError());
    

}

void destroy() {
    hipFree(gpuDataCPU->distanceData);
    free(gpuDataCPU);
    hipFree(gpuDataLOCAL);
}